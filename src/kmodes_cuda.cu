#include "hip/hip_runtime.h"
#include "global.h" 
#include "sequence.h"

extern "C" {
  #include "kmodes.h"
}

#include <string.h>
#include <hip/hip_runtime.h>

#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) exit(code);
  }
}

__device__ unsigned int cuda_delta = 0;



__global__ void update_centroids(const sequence_t *data, sequence_t *centroids, unsigned int * tmp_centroidCount, unsigned int numClusters) {
  int i = blockIdx.x * blockDim.x  + threadIdx.x;

  if (i < numClusters) {
    sequence_t seq = make_ulong3(0,0,0);
    unsigned int *tmp_centroid = &tmp_centroidCount[i* BIT_SIZE_OF(sequence_t)];
    for (int j=0;j<SEQ_DIM_BITS_SIZE;j+=4)
    {

      // bits tmp_centroid[0] is less significative bit from sequence_t
      // bits tmp_centroid[0] = z << 0
      unsigned int *bitCountX = &tmp_centroid[j + (SEQ_DIM_BITS_SIZE * 2)];
      unsigned int *bitCountY = &tmp_centroid[j + SEQ_DIM_BITS_SIZE];
      unsigned int *bitCountZ = &tmp_centroid[j];

      unsigned long int mask = maskForMode(bitCountX[0],bitCountX[1],bitCountX[2],bitCountX[3]);
      seq.x |= (mask << j);
      mask = maskForMode(bitCountY[0],bitCountY[1],bitCountY[2],bitCountY[3]);
      seq.y |= (mask << j);
      mask = maskForMode(bitCountZ[0],bitCountZ[1],bitCountZ[2],bitCountZ[3]);
      seq.z |= (mask << j);
    }
    centroids[i] = seq;
  }
}

__global__ void kmeans_kernel(const sequence_t *data, const sequence_t *centroids, int * membership, unsigned int * tmp_centroidCount, unsigned int n, unsigned int numClusters) {

  int index = blockIdx.x * blockDim.x  + threadIdx.x;
  if (index < n){

    unsigned int min_distance = UINT_MAX;
    long nearest = -1;

    for (int i = 0; i < numClusters; i++){
      sequence_t centroid = centroids[i];
      unsigned int distance = __popcll(centroid.x ^ data[index].x) +
      __popcll(centroid.y ^ data[index].y) +
      __popcll(centroid.z ^ data[index].z);
      if(distance < min_distance) {
        nearest = i;
        min_distance = distance;
      }
    }

    if(membership[index] != nearest) {
      membership[index]=nearest;
      atomicInc(&cuda_delta,n*2);
    }

    unsigned int *tmp_centroid = &tmp_centroidCount[membership[index] * BIT_SIZE_OF(sequence_t)];
    for (unsigned int j=0;j<SEQ_DIM_BITS_SIZE;j++)  {
      // bits tmp_centroid[0] is less significative bit from sequence_t
      // bits tmp_centroid[0] = z << 0
      unsigned long int mask = 1;
      if (data[index].z & (mask << j)){
        atomicInc(&tmp_centroid[j],n);
      }
      if (data[index].y & (mask << j)){
        atomicInc(&tmp_centroid[SEQ_DIM_BITS_SIZE + j],n);
      }
      if (data[index].x & (mask << j)){
        atomicInc(&tmp_centroid[(2 *SEQ_DIM_BITS_SIZE)+j],n);
      }
    }
  }
}


void checkCudaMemory() {
  // show memory usage of GPU

  size_t free_byte ;
  size_t total_byte;
  checkCuda(hipMemGetInfo( &free_byte, &total_byte ));
  double free_db = (double)free_byte ;
  double total_db = (double)total_byte ;
  double used_db = total_db - free_db ;
  printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
  used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}


extern "C" kmodes_result_t kmodes(kmodes_input_t input) {
  printf("%s\n", "Execution cuda kmodes");
  //checkCudaMemory();

  unsigned int delta=0; //Number of objects has diverged in current iteration

  size_t clusters = input.number_of_clusters;
  size_t data_size = input.data_size;
  sequence_t *data = input.data;
  int* label = (int*)calloc(data_size, sizeof(int));
  sequence_t *centroids = (sequence_t*)calloc(clusters, sizeof(sequence_t));


  // cuda variables
  sequence_t * cuda_data = NULL;
  sequence_t * cuda_centroids = NULL;
  int *cuda_membership = NULL;
  unsigned int *cuda_tmp_centroidCount = NULL;

  //memory arrays size
  const unsigned int cuda_tmp_centroidCountSize = clusters * BIT_SIZE_OF(sequence_t) *sizeof(unsigned int);

  const unsigned int threadsPerBlock = 128;
  const unsigned int numBlocks = (data_size + threadsPerBlock - 1) / threadsPerBlock;
  const unsigned int numBlocks2 = (clusters + threadsPerBlock - 1) / threadsPerBlock;

  for(unsigned int i = 0;i < clusters;i++) {
    int h = i * data_size / clusters;
    centroids[i] = make_ulong3(data[h].x,data[h].y,data[h].z);
  }

  memset (label,-1,data_size * sizeof(int));

  checkCuda(hipMalloc(&cuda_data, data_size * sizeof(sequence_t)));
  checkCuda(hipMalloc(&cuda_centroids, clusters * sizeof(sequence_t)));
  checkCuda(hipMalloc(&cuda_membership, data_size * sizeof(int)));
  checkCuda(hipMalloc(&cuda_tmp_centroidCount,cuda_tmp_centroidCountSize));
  hipDeviceSynchronize();

  checkCuda(hipMemcpy(cuda_data,data, data_size *sizeof(sequence_t) , hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(cuda_centroids, centroids, clusters *sizeof(sequence_t) , hipMemcpyHostToDevice));
  checkCuda(hipMemset(cuda_membership,-1,data_size * sizeof(int)));
  hipDeviceSynchronize();

  printf("Num blocks: %d Num threads: %d total %d\n",numBlocks,threadsPerBlock,threadsPerBlock * numBlocks);

  do {
    delta = 0;

    checkCuda(hipMemset(cuda_tmp_centroidCount,0,cuda_tmp_centroidCountSize));
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(cuda_delta), &delta,sizeof(unsigned int),0,hipMemcpyHostToDevice));
    hipDeviceSynchronize();

    // first kernel
    kmeans_kernel <<< numBlocks,threadsPerBlock>>>(cuda_data,
      cuda_centroids,
      cuda_membership,
      cuda_tmp_centroidCount,
      data_size,
      clusters);

    checkCuda(hipMemcpy(label,cuda_membership, data_size *sizeof(int), hipMemcpyDeviceToHost));
    checkCuda(hipGetLastError());
    hipDeviceSynchronize();

    // second kernel
    update_centroids<<<numBlocks2,threadsPerBlock>>>(cuda_data,
      cuda_centroids,
      cuda_tmp_centroidCount,
      clusters);

    checkCuda(hipGetLastError());
    hipDeviceSynchronize();
    checkCuda(hipMemcpyFromSymbol(&delta,HIP_SYMBOL(cuda_delta),sizeof(unsigned int)));
    hipDeviceSynchronize();

  } while(delta > 0);

  // copy output
  checkCuda(hipMemcpy(label,cuda_membership, data_size *sizeof(int) , hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(centroids,cuda_centroids, clusters *sizeof(sequence_t) , hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  // free cuda memory
  checkCuda(hipFree(cuda_data));
  checkCuda(hipFree(cuda_centroids));
  checkCuda(hipFree(cuda_membership));
  checkCuda(hipFree(cuda_tmp_centroidCount));
  checkCuda(hipDeviceReset());

  kmodes_result_t result = {
    label,
    centroids
  };
  return result;
}

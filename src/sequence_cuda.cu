#include "hip/hip_runtime.h"
//
//  sequence.c
//  TD
//
//  Created by Guilherme Torres on 9/7/13.
//  Copyright (c) 2013 Guilherme Torres. All rights reserved.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "sequence.h"

static size_t const sequence_part1_size = sizeof(uint64_t) * 8;
static size_t const sequence_part2_size = sequence_part1_size * 2;


__OFFLOAD__MODIFIER__ inline void print_sequence(sequence_t seq) {
  const char* string = to_string(seq);
  printf("%s\n", string);
  free(string);
}

__OFFLOAD__MODIFIER__ inline int dist_sequence(sequence_t seq1,sequence_t seq2) {
  unsigned long int xDiff = seq1.x ^ seq2.x;
  unsigned long int yDiff = seq1.y ^ seq2.y;
  unsigned long int zDiff = seq1.z ^ seq2.z;
  return __builtin_popcountl(xDiff) + __builtin_popcountl(yDiff) + __builtin_popcountl(zDiff);
}

__OFFLOAD__MODIFIER__ inline sequence_t copy_sequence(sequence_t seq) {
  sequence_t sequence = { seq.x,seq.y,seq.z };
  return sequence;
}


__OFFLOAD__MODIFIER__ inline bool get_bit(sequence_t sequence, size_t position) {

  size_t part = (position < sequence_part1_size) ? sequence.x :
    (position < sequence_part2_size) ? sequence.y : sequence.z;
  size_t part_position = position % sequence_part1_size;
  uint64_t mask = 1;
  return part & (mask << part_position);
}

__OFFLOAD__MODIFIER__ sequence_t sum_sequence(sequence_t seq1,sequence_t seq2) {
  sequence_t sequence = { seq1.x + seq2.x,seq1.y + seq2.y,seq1.z + seq2.z };
  return sequence;
}

__OFFLOAD__MODIFIER__ sequence_t inline div_sequence(sequence_t seq1, unsigned long div) {
  sequence_t sequence = { seq1.x / div , seq1.y / div, seq1.z / div };
  return sequence;
}

__OFFLOAD__MODIFIER__ inline sequence_t make_sequence(uint64_t x, uint64_t y, uint64_t z) {
  return make_ulong3(x, y, z);
}

make_sequence
